
#include <hip/hip_runtime.h>
void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
   // set your block dimensions and grid dimensions here
   gridDim.x = n / blockDim.x/4;
   gridDim.y = n / blockDim.y/4;
   if(n % blockDim.x != 0)
    gridDim.x++;
   if(n % blockDim.y != 0)
        gridDim.y++;    
   
   //blockDim.y = blockDim.y/8;
}
